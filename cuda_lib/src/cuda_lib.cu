#include "hip/hip_runtime.h"
#include <cstdio>
#include "cuda_lib.h"
#include <hip/hip_runtime.h>

__global__ void CudaLibFuncImpl() { printf("Hello my CUDA library\n"); }

void CudaLibFunc()
{
    CudaLibFuncImpl<<<1, 1>>>();
    hipDeviceSynchronize();
}
